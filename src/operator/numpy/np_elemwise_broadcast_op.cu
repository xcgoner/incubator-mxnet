#include "hip/hip_runtime.h"
/*
 * Licensed to the Apache Software Foundation (ASF) under one
 * or more contributor license agreements.  See the NOTICE file
 * distributed with this work for additional information
 * regarding copyright ownership.  The ASF licenses this file
 * to you under the Apache License, Version 2.0 (the
 * "License"); you may not use this file except in compliance
 * with the License.  You may obtain a copy of the License at
 *
 *   http://www.apache.org/licenses/LICENSE-2.0
 *
 * Unless required by applicable law or agreed to in writing,
 * software distributed under the License is distributed on an
 * "AS IS" BASIS, WITHOUT WARRANTIES OR CONDITIONS OF ANY
 * KIND, either express or implied.  See the License for the
 * specific language governing permissions and limitations
 * under the License.
 */

/*!
 *  Copyright (c) 2019 by Contributors
 * \file np_elemwise_broadcast_op.cu
 * \brief GPU Implementation of basic functions for elementwise binary broadcast operator.
 */

#include "./np_elemwise_broadcast_op.h"

namespace mxnet {
namespace op {

NNVM_REGISTER_OP(_npi_add)
#ifndef _WIN32
.set_attr<FCompute>(
  "FCompute<gpu>",
  NumpyBinaryBroadcastComputeWithBool<gpu, op::mshadow_op::plus, op::mshadow_op::mixed_plus,
                                      op::mshadow_op::mixed_plus>);
#else
.set_attr<FCompute>(
  "FCompute<gpu>",
  NumpyBinaryBroadcastComputeWithBool<gpu, op::mshadow_op::plus>);
#endif

NNVM_REGISTER_OP(_npi_subtract)
#ifndef _WIN32
.set_attr<FCompute>(
  "FCompute<gpu>",
  NumpyBinaryBroadcastCompute<gpu, op::mshadow_op::minus, op::mshadow_op::mixed_minus,
                              op::mshadow_op::mixed_rminus>);
#else
.set_attr<FCompute>(
  "FCompute<gpu>",
  NumpyBinaryBroadcastCompute<gpu, op::mshadow_op::minus>);
#endif

NNVM_REGISTER_OP(_npi_multiply)
#ifndef _WIN32
.set_attr<FCompute>(
  "FCompute<gpu>",
  NumpyBinaryBroadcastComputeWithBool<gpu, op::mshadow_op::mul, op::mshadow_op::mixed_mul,
                                      op::mshadow_op::mixed_mul>);
#else
.set_attr<FCompute>(
  "FCompute<gpu>",
  NumpyBinaryBroadcastComputeWithBool<gpu, op::mshadow_op::mul>);
#endif

NNVM_REGISTER_OP(_npi_mod)
.set_attr<FCompute>("FCompute<gpu>", BinaryBroadcastCompute<gpu, mshadow_op::mod>);

NNVM_REGISTER_OP(_npi_power)
.set_attr<FCompute>("FCompute<gpu>", BinaryBroadcastCompute<gpu, mshadow_op::power>);

NNVM_REGISTER_OP(_npi_copysign)
.set_attr<FCompute>("FCompute<gpu>", BinaryBroadcastCompute<gpu, mshadow_op::copysign>);

NNVM_REGISTER_OP(_npi_lcm)
.set_attr<FCompute>("FCompute<gpu>", BinaryBroadcastCompute<gpu, mshadow_op::lcm>);

NNVM_REGISTER_OP(_backward_npi_copysign)
.set_attr<FCompute>("FCompute<gpu>", BinaryBroadcastBackwardUseIn<gpu, mshadow_op::copysign_grad,
                                                                  mshadow_op::copysign_rgrad>);

NNVM_REGISTER_OP(_npi_arctan2)
.set_attr<FCompute>("FCompute<gpu>", BinaryBroadcastCompute<gpu, mshadow_op::arctan2>);

NNVM_REGISTER_OP(_backward_npi_arctan2)
.set_attr<FCompute>("FCompute<gpu>", BinaryBroadcastBackwardUseIn<gpu, mshadow_op::arctan2_grad,
                                                                  mshadow_op::arctan2_rgrad>);
NNVM_REGISTER_OP(_npi_hypot)
.set_attr<FCompute>("FCompute<gpu>", BinaryBroadcastCompute<gpu, mshadow_op::hypot>);

NNVM_REGISTER_OP(_backward_npi_hypot)
.set_attr<FCompute>("FCompute<gpu>", BinaryBroadcastBackwardUseIn<gpu, mshadow_op::hypot_grad_left,
                                                                  mshadow_op::hypot_grad_right>);

NNVM_REGISTER_OP(_npi_add_scalar)
.set_attr<FCompute>("FCompute<gpu>", BinaryScalarOp::Compute<gpu, op::mshadow_op::plus>);

NNVM_REGISTER_OP(_npi_subtract_scalar)
.set_attr<FCompute>("FCompute<gpu>", BinaryScalarOp::Compute<gpu, op::mshadow_op::minus>);

NNVM_REGISTER_OP(_npi_rsubtract_scalar)
.set_attr<FCompute>("FCompute<gpu>", BinaryScalarOp::Compute<gpu, mshadow_op::rminus>);

NNVM_REGISTER_OP(_npi_multiply_scalar)
.set_attr<FCompute>("FCompute<gpu>", BinaryScalarOp::Compute<gpu, op::mshadow_op::mul>);

NNVM_REGISTER_OP(_npi_mod_scalar)
.set_attr<FCompute>("FCompute<gpu>", BinaryScalarOp::Compute<gpu, mshadow_op::mod>);

NNVM_REGISTER_OP(_npi_rmod_scalar)
.set_attr<FCompute>("FCompute<gpu>", BinaryScalarOp::Compute<gpu, mshadow_op::rmod>);

NNVM_REGISTER_OP(_npi_power_scalar)
.set_attr<FCompute>("FCompute<gpu>", BinaryScalarOp::Compute<gpu, mshadow_op::power>);

NNVM_REGISTER_OP(_npi_rpower_scalar)
.set_attr<FCompute>("FCompute<gpu>", BinaryScalarOp::Compute<gpu, mshadow_op::rpower>);

NNVM_REGISTER_OP(_npi_copysign_scalar)
.set_attr<FCompute>("FCompute<gpu>", BinaryScalarOp::Compute<gpu, mshadow_op::copysign>);

NNVM_REGISTER_OP(_npi_rcopysign_scalar)
.set_attr<FCompute>("FCompute<gpu>", BinaryScalarOp::Compute<gpu, mshadow_op::rcopysign>);

NNVM_REGISTER_OP(_backward_npi_copysign_scalar)
.set_attr<FCompute>("FCompute<gpu>",
                    BinaryScalarOp::Backward<gpu, mshadow_op::copysign_grad>);

NNVM_REGISTER_OP(_backward_npi_rcopysign_scalar)
.set_attr<FCompute>("FCompute<gpu>",
                    BinaryScalarOp::Backward<gpu, mshadow_op::rcopysign_grad>);

NNVM_REGISTER_OP(_npi_arctan2_scalar)
.set_attr<FCompute>("FCompute<gpu>", BinaryScalarOp::Compute<gpu, mshadow_op::arctan2>);

NNVM_REGISTER_OP(_backward_npi_arctan2_scalar)
.set_attr<FCompute>("FCompute<gpu>", BinaryScalarOp::Compute<gpu, mshadow_op::arctan2_grad>);

NNVM_REGISTER_OP(_npi_rarctan2_scalar)
.set_attr<FCompute>("FCompute<gpu>", BinaryScalarOp::Compute<gpu, mshadow_op::rarctan2>);

NNVM_REGISTER_OP(_backward_npi_rarctan2_scalar)
.set_attr<FCompute>("FCompute<gpu>", BinaryScalarOp::Compute<gpu, mshadow_op::rarctan2_grad>);

NNVM_REGISTER_OP(_npi_lcm_scalar)
.set_attr<FCompute>("FCompute<gpu>", BinaryScalarOp::Compute<gpu, mshadow_op::lcm>);

NNVM_REGISTER_OP(_npi_ldexp)
.set_attr<FCompute>("FCompute<gpu>", BinaryBroadcastCompute<gpu, mshadow_op::ldexp>);

NNVM_REGISTER_OP(_npi_ldexp_scalar)
.set_attr<FCompute>("FCompute<gpu>", BinaryScalarOp::Compute<gpu, mshadow_op::ldexp>);

NNVM_REGISTER_OP(_npi_rldexp_scalar)
.set_attr<FCompute>("FCompute<gpu>", BinaryScalarOp::Compute<gpu, mshadow_op::rldexp>);

NNVM_REGISTER_OP(_backward_npi_ldexp)
.set_attr<FCompute>("FCompute<gpu>", BinaryBroadcastBackwardUseIn<gpu, mshadow_op::ldexp_grad,
                                                                  mshadow_op::ldexp_rgrad>);

NNVM_REGISTER_OP(_backward_npi_ldexp_scalar)
.set_attr<FCompute>("FCompute<gpu>", BinaryScalarOp::Backward<gpu, mshadow_op::ldexp_grad>);

NNVM_REGISTER_OP(_backward_npi_rldexp_scalar)
.set_attr<FCompute>("FCompute<gpu>", BinaryScalarOp::Backward<gpu, mshadow_op::rldexp_grad>);

}  // namespace op
}  // namespace mxnet
